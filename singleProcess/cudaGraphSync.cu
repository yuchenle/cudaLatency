#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <vector>

#include "../cudaErr.h"

__global__ void dummy_kernel()
{
}

int main()
{
  hipGraph_t graph;
  hipStream_t streamForGraph;
  gpuErrchk (hipGraphCreate (&graph, 0));
  gpuErrchk (hipStreamCreate (&streamForGraph));

  hipGraphNode_t dummyNode1, dummyNode2;

  std::vector<hipGraphNode_t> nodeDep;

  // adding first dummy node to the graph
  hipKernelNodeParams kernelNodeParams = {0};

  kernelNodeParams.func = (void *) dummy_kernel;
  kernelNodeParams.gridDim = dim3 (1, 1, 1); 
  kernelNodeParams.blockDim = dim3 (1, 1, 1); 
  kernelNodeParams.sharedMemBytes = 0;
  kernelNodeParams.extra = NULL;
  kernelNodeParams.kernelParams = (void **)NULL;
  gpuErrchk (hipGraphAddKernelNode (&dummyNode1, graph, NULL, 0, &kernelNodeParams));

  nodeDep.push_back (dummyNode1);

  // adding second dummy node
  // memset (&kernelNodeParams, 0, sizeof (kernelNodeParams));
  // kernelNodeParams.func = (void *)dummy_kernel;
  // kernelNodeParams.gridDim = dim3 (1,1,1);
  // kernelNodeParams.blockDim = dim3 (1,1,1);
  // kernelNodeParams.sharedMemBytes = 0;
  // kernelNodeParams.extra = NULL;
  gpuErrchk (
	hipGraphAddKernelNode (&dummyNode2, graph, nodeDep.data(), 
				nodeDep.size(), &kernelNodeParams));

  hipGraphNode_t *nodes = NULL;
  size_t numNodes = 0;
  gpuErrchk (hipGraphGetNodes (graph, nodes, &numNodes));
  printf ("number of nodes in graph is %zu\n", numNodes);

  hipGraphExec_t graphExec;
  gpuErrchk (hipGraphInstantiate (&graphExec, graph, NULL, NULL, 0));

  struct timespec stamp, previous_stamp;
  clock_gettime (CLOCK_MONOTONIC, &stamp);
  double wtime;
  
  for (int i = 0; i < 1000000; i++)
  {
    gpuErrchk (hipGraphLaunch (graphExec, streamForGraph));
    gpuErrchk (hipStreamSynchronize (streamForGraph));
    memcpy (&previous_stamp, &stamp, sizeof (struct timespec));
    clock_gettime (CLOCK_MONOTONIC, &stamp);
    wtime = (stamp.tv_sec - previous_stamp.tv_sec) * 1000000 + (stamp.tv_nsec - previous_stamp.tv_nsec) / 1000;
    printf ("%.4f \n", wtime);
  }
  
}
