#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "../cudaErr.h"

__global__ void dummy_kernel()
{
}

int main()
{
  struct timespec stamp, previous_stamp;
  clock_gettime (CLOCK_MONOTONIC, &stamp);
  double wtime;
  hipStream_t st1;
  gpuErrchk (hipStreamCreate (&st1));
  
  for (int i = 0; i < 1000000; i++)
  {
    dummy_kernel <<<1, 1, 0, st1>>>();
    dummy_kernel <<<1, 1, 0, st1>>>();
    gpuErrchk (hipStreamSynchronize(st1));
    memcpy (&previous_stamp, &stamp, sizeof (struct timespec));
    clock_gettime (CLOCK_MONOTONIC, &stamp);
    wtime = (stamp.tv_sec - previous_stamp.tv_sec) * 1000000 + (stamp.tv_nsec - previous_stamp.tv_nsec) / 1000;
    printf ("%.4f \n", wtime);
  }
  
}
