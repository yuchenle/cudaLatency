#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/ipc.h>
#include <sys/shm.h>
#include <errno.h>
#include <signal.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "../cudaErr.h"
#include "const.h"
#include "Sender.hpp"

// global variables to be initialized
int shmid;
TYPE *ptr;
hipStream_t RT_stream_g;
hipStream_t callBack_stream_g;
hipEvent_t event_g;

void init_shm (TYPE *ptr)
{
  for (int i = 0; i < SIZE; i++)
    ptr->ready[i] = WAIT_VALUE;

  gpuErrchk (hipStreamCreate (&RT_stream_g));
  gpuErrchk (hipStreamCreate (&callBack_stream_g));

  memcpy (&(ptr->RT_stream), &RT_stream_g, sizeof(hipStream_t)); 
  memcpy (&(ptr->callBackStream), &callBack_stream_g, sizeof(hipStream_t)); 


  // gpuErrchk (hipStreamCreate (&(ptr->RT_stream)));
  // gpuErrchk (hipStreamCreate (&(ptr->callBackStream)));
  // ptr->RT_stream = RT_stream_g;
  // ptr->callBackStream = &callBack_stream_g;
  // ptr->memHandle = (hipIpcMemHandle_t *)malloc (sizeof (hipIpcMemHandle_t));
  // ptr->eventHandle = (hipIpcEventHandle_t *)malloc (sizeof (hipIpcEventHandle_t));
}

void init ()
{
  // FILE to key
  key_t key = ftok (FILENAME, FILEID);
  if (key == -1) 
  {
    printf ("ftok failed, errno = %s\n", strerror(errno));
    exit (-1);
  }

  // getting SHM id
  printf (" size of shm is %zu\n", sizeof(TYPE));
  shmid = shmget (key, sizeof (TYPE), 0666|IPC_CREAT);
  if (shmid == -1) 
  {
    printf ("shmget failed, errno = %s\n", strerror(errno));
    exit (-1);
  }

  // attach the SHM to this process
  ptr = (TYPE *) shmat (shmid, (void *)0, 0); 

  init_shm (ptr);
}

__global__ void init_kernel (GPU_DATA_TYPE *d_data)
{
  int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;

  while (threadId < GPU_SIZE)
  {
    d_data[threadId] = 1;
    threadId += blockDim.x * gridDim.x;
  }
}

void sigInt_handler (int sig)
{
  if (sig == SIGINT)
  {
    printf ("sender received SIGINT, calling hipProfilerStop before exiting\n");
    gpuErrchk (hipProfilerStop ());
    exit (0);
  }
}

int main()
{
  init ();
  printf ("ptr is %p\n", (void *)ptr);

  gpuErrchk (hipProfilerStart ());

  GPU_DATA_TYPE *d_a;
  gpuErrchk (hipMalloc (&d_a, sizeof(GPU_DATA_TYPE) * GPU_SIZE));

  init_kernel<<<128,1024>>> (d_a);

  Sender *sender = new Sender(shmid, ptr, d_a);

  hipIpcMemHandle_t handle;
  gpuErrchk (hipIpcGetMemHandle (&handle, d_a));
  
  memcpy (&(ptr->memHandle), &handle, sizeof (hipIpcMemHandle_t));

  sender->set_GPUIPC_handle (&handle);

  // set the signal handling function
  if (signal (SIGINT, sigInt_handler) == SIG_ERR)
  {
    printf ("cannot handle SIGINT\n");
    exit(-1);
  }

  // main loop
  while (true)
  // for (int i = 0;i < 1000; i++)
  {
    sender->update ();
    sender->wait ();
    sender->process ();
    sender->notify ();
    // usleep (20);
  }
}
